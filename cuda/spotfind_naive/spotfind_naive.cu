#include "hip/hip_runtime.h"
/**
 * Basic Summing with CUDA
 * 
 * Uses h5read to loop over all images, calculates a pixel sum in host
 * and GPU, and compares the results.
 * 
 * Demonstrates using h5read and GPU reduction.
 * 
 */

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <fmt/core.h>

#include <array>
#include <cassert>
#include <memory>

#include "common.hpp"
#include "h5read.h"

namespace cg = cooperative_groups;

using namespace fmt;

using pixel_t = H5Read::image_type;

int main(int argc, char **argv) {
    // Parse arguments and get our H5Reader
    auto parser = CUDAArgumentParser();
    parser.add_h5read_arguments();
    auto args = parser.parse_args(argc, argv);

    auto reader = args.file.empty() ? H5Read() : H5Read(args.file);

    int height = reader.image_shape()[0];
    int width = reader.image_shape()[1];

    // Work out how many blocks this is
    dim3 thread_block_size{32, 16};
    dim3 blocks_dims{
      static_cast<unsigned int>(ceilf((float)width / thread_block_size.x)),
      static_cast<unsigned int>(ceilf((float)height / thread_block_size.y))};
    const int num_threads_per_block = thread_block_size.x * thread_block_size.y;
    const int num_blocks = blocks_dims.x * blocks_dims.y;
    print("Image:   {:4d} x {:4d} = {} px\n", width, height, width * height);
    print("Threads: {:4d} x {:<4d} = {}\n",
          thread_block_size.x,
          thread_block_size.y,
          num_threads_per_block);
    print("Blocks:  {:4d} x {:<4d} = {}\n", blocks_dims.x, blocks_dims.y, num_blocks);
}
