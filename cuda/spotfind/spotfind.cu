#include "hip/hip_runtime.h"
/**
 * Basic Naive Kernel
 * 
 * Does spotfinding in-kernel, without in-depth performance tweaking.
 * 
 */

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <fmt/core.h>

#include <array>
#include <cassert>
#include <chrono>
#include <memory>
#include <utility>

#include "common.hpp"
#include "h5read.h"
#include "standalone.h"

namespace cg = cooperative_groups;

using namespace fmt;

using pixel_t = H5Read::image_type;

/// One-direction width of kernel. Total kernel span is (K_W * 2 + 1)
constexpr int KERNEL_WIDTH = 3;
/// One-direction height of kernel. Total kernel span is (K_H * 2 + 1)
constexpr int KERNEL_HEIGHT = 3;

__global__ void do_spotfinding_naive(pixel_t *image,
                                     size_t image_pitch,
                                     uint8_t *mask,
                                     size_t mask_pitch,
                                     int width,
                                     int height,
                                     int *result_sum,
                                     size_t *result_sumsq,
                                     uint8_t *result_n,
                                     uint8_t *result_strong) {
    auto block = cg::this_thread_block();
    // auto warp = cg::tiled_partition<32>(block);
    // int warpId = warp.meta_group_rank();
    // int lane = warp.thread_rank();

    uint sum = 0;
    size_t sumsq = 0;
    uint8_t n = 0;

    int x = block.group_index().x * block.group_dim().x + block.thread_index().x;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y;

    // Don't calculate for masked pixels
    bool px_is_valid = mask[y * mask_pitch + x] != 0;
    pixel_t this_pixel = image[y * image_pitch + x];

    if (px_is_valid) {
        for (int row = max(0, y - KERNEL_HEIGHT);
             row < min(y + KERNEL_HEIGHT + 1, height);
             ++row) {
            int row_offset = image_pitch * row;
            int mask_offset = mask_pitch * row;
            for (int col = max(0, x - KERNEL_WIDTH);
                 col < min(x + KERNEL_WIDTH + 1, width);
                 ++col) {
                pixel_t pixel = image[row_offset + col];
                uint8_t mask_pixel = mask[mask_offset + col];
                if (mask_pixel) {
                    sum += pixel;
                    sumsq += pixel * pixel;
                    n += 1;
                }
            }
        }
    }

    if (x < width && y < height) {
        // result_sum[x + image_pitch * y] = sum;
        // result_sumsq[x + image_pitch * y] = sumsq;
        // result_n[x + mask_pitch * y] = n;

        // Calculate the thresholding
        if (px_is_valid) {
            constexpr float n_sig_s = 3.0f;
            constexpr float n_sig_b = 6.0f;

            float sum_f = static_cast<float>(sum);
            float sumsq_f = static_cast<float>(sumsq);

            float mean = sum_f / n;
            float variance = (n * sumsq_f - (sum_f * sum_f)) / (n * (n - 1));
            float dispersion = variance / mean;
            float background_threshold = 1 + n_sig_b * sqrt(2.0f / (n - 1));
            bool not_background = dispersion > background_threshold;
            float signal_threshold = mean + n_sig_s * sqrt(mean);
            bool is_signal = this_pixel > signal_threshold;
            bool is_strong_pixel = not_background && is_signal;
            result_strong[x + mask_pitch * y] = is_strong_pixel;
        } else {
            result_strong[x + mask_pitch * y] = 0;
        }
    }
}

int main(int argc, char **argv) {
    // Parse arguments and get our H5Reader
    auto parser = CUDAArgumentParser();
    parser.add_h5read_arguments();
    auto args = parser.parse_args(argc, argv);

    auto reader = args.file.empty() ? H5Read() : H5Read(args.file);

    int height = reader.image_shape()[0];
    int width = reader.image_shape()[1];

    // Work out how many blocks this is
    dim3 thread_block_size{32, 16};
    dim3 blocks_dims{
      static_cast<unsigned int>(ceilf((float)width / thread_block_size.x)),
      static_cast<unsigned int>(ceilf((float)height / thread_block_size.y))};
    const int num_threads_per_block = thread_block_size.x * thread_block_size.y;
    const int num_blocks = blocks_dims.x * blocks_dims.y;
    print("Image:   {:4d} x {:4d} = {} px\n", width, height, width * height);
    print("Threads: {:4d} x {:<4d} = {}\n",
          thread_block_size.x,
          thread_block_size.y,
          num_threads_per_block);
    print("Blocks:  {:4d} x {:<4d} = {}\n", blocks_dims.x, blocks_dims.y, num_blocks);

    // Create a host memory area to read the image into
    // auto host_image = std::make_unique<pixel_t[]>(width * height);
    auto host_image = make_cuda_pinned_malloc<pixel_t>(width * height);

    // Device-side pitched storage for image data
    auto [dev_image, device_pitch] = make_cuda_pitched_malloc<pixel_t>(width, height);
    auto [dev_mask, device_mask_pitch] =
      make_cuda_pitched_malloc<uint8_t>(width, height);
    print("Allocated device memory. Pitch = {} vs naive {}\n", device_pitch, width);

    // Managed memory areas for results
    auto result_sum = make_cuda_managed_malloc<int>(device_pitch * height);
    auto result_sumsq = make_cuda_managed_malloc<size_t>(device_pitch * height);
    auto result_n = make_cuda_managed_malloc<uint8_t>(device_mask_pitch * height);
    auto result_strong = make_cuda_managed_malloc<uint8_t>(device_mask_pitch * height);
    // Make sure to clear these completely
    hipMemset(result_sum.get(), 0, sizeof(int) * device_pitch * height);
    hipMemset(result_sumsq.get(), 0, sizeof(size_t) * device_pitch * height);
    hipMemset(result_n.get(), 0, sizeof(uint8_t) * device_mask_pitch * height);
    hipMemset(result_strong.get(), 0, sizeof(uint8_t) * device_mask_pitch * height);
    hipDeviceSynchronize();
    cuda_throw_error();

    CudaEvent pre_load, start, memcpy, kernel, all;

    size_t mask_sum = 0;
    if (reader.get_mask()) {
        mask_sum = 0;
        for (size_t i = 0; i < width * height; ++i) {
            if (reader.get_mask().value()[i]) {
                mask_sum += 1;
            }
        }
        start.record();
        hipMemcpy2D(dev_mask.get(),
                     device_mask_pitch,
                     reader.get_mask()->data(),
                     width,
                     width,
                     height,
                     hipMemcpyHostToDevice);
        cuda_throw_error();
    } else {
        mask_sum = width * height;
        start.record();
        hipMemset(dev_mask.get(), 1, device_mask_pitch * height);
        cuda_throw_error();
    }
    memcpy.record();
    memcpy.synchronize();

    float memcpy_time = memcpy.elapsed_time(start);
    print("Uploaded mask ({:.2f} Mpx) in {:.2f} ms ({:.1f} GBps)\n",
          static_cast<float>(mask_sum) / 1e6,
          memcpy_time,
          GBps(memcpy_time, width * height));

    print("\nProcessing {} Images\n\n", reader.get_number_of_images());
    auto spotfinder = StandaloneSpotfinder(width, height);

    for (size_t image_id = 0; image_id < reader.get_number_of_images(); ++image_id) {
        if (args.image_number.has_value() && args.image_number.value() != image_id) {
            continue;
        }

        print("Image {}:\n", image_id);
        pre_load.record();
        pre_load.synchronize();

        reader.get_image_into(image_id, host_image.get());

        // Copy data to GPU
        // Copy the image to GPU
        start.record();
        hipMemcpy2D(dev_image.get(),
                     device_pitch * sizeof(pixel_t),
                     host_image.get(),
                     width * sizeof(pixel_t),
                     width * sizeof(pixel_t),
                     height,
                     hipMemcpyHostToDevice);
        memcpy.record();
        hipDeviceSynchronize();
        cuda_throw_error();

        do_spotfinding_naive<<<blocks_dims, thread_block_size>>>(dev_image.get(),
                                                                 device_pitch,
                                                                 dev_mask.get(),
                                                                 device_mask_pitch,
                                                                 width,
                                                                 height,
                                                                 result_sum.get(),
                                                                 result_sumsq.get(),
                                                                 result_n.get(),
                                                                 result_strong.get());
        kernel.record();
        all.record();
        cuda_throw_error();
        hipDeviceSynchronize();

        print("    Read Time: \033[1m{:6.2f}\033[0m ms \033[37m({:.1f} GBps)\033[0m\n",
              start.elapsed_time(pre_load),
              GBps<pixel_t>(start.elapsed_time(pre_load), width * height));
        print("  Upload Time: \033[1m{:6.2f}\033[0m ms \033[37m({:.1f} GBps)\033[0m\n",
              memcpy.elapsed_time(start),
              GBps<pixel_t>(memcpy.elapsed_time(start), width * height));
        print("  Kernel Time: \033[1m{:6.2f}\033[0m ms\n", kernel.elapsed_time(memcpy));
        print("               ════════\n");
        print("        Total: \033[1m{:6.2f}\033[0m ms ({:.1f} GBps)\n",
              all.elapsed_time(pre_load),
              GBps<pixel_t>(all.elapsed_time(pre_load), width * height));

        auto strong =
          count_nonzero(result_strong.get(), width, height, device_mask_pitch);
        print("       Strong: {} px\n", strong);

        auto start_time = std::chrono::high_resolution_clock::now();
        size_t mismatch_x = 0, mismatch_y = 0;

        auto converted_image =
          std::vector<double>{host_image.get(), host_image.get() + width * height};
        auto dials_strong = spotfinder.standard_dispersion(
          converted_image, reader.get_mask().value_or(span<uint8_t>{}));
        auto end_time = std::chrono::high_resolution_clock::now();
        size_t dials_results = count_nonzero(dials_strong, width, height, width);

        print("        Dials: {} px\n", dials_results);
        bool validation_matches = compare_results(dials_strong.data(),
                                                  width,
                                                  result_strong.get(),
                                                  device_mask_pitch,
                                                  width,
                                                  height,
                                                  &mismatch_x,
                                                  &mismatch_y);
        float validation_time_ms =
          std::chrono::duration_cast<std::chrono::duration<double>>(end_time
                                                                    - start_time)
            .count()
          * 1000;

        if (validation_matches) {
            print("     Compared: \033[32mMatch\033[0m in {:.0f} ms\n",
                  validation_time_ms);
        } else {
            print("     Compared: \033[1;31mMismatch\033[0m in {:.0f} ms\n",
                  validation_time_ms);
            mismatch_x = max(static_cast<int>(mismatch_x) - 8, 0);
            mismatch_y = max(static_cast<int>(mismatch_y) - 8, 0);
            print("Data:\n");
            draw_image_data(host_image, mismatch_x, mismatch_y, 16, 16, width, height);
            print("Strong From DIALS:\n");
            draw_image_data(
              dials_strong, mismatch_x, mismatch_y, 16, 16, width, height);
            print("Strong From kernel:\n");
            draw_image_data(
              result_strong, mismatch_x, mismatch_y, 16, 16, device_mask_pitch, height);
            // print("Resultant N:\n");
            print("Sum From kernel:\n");
            draw_image_data(
              result_sum, mismatch_x, mismatch_y, 16, 16, device_pitch, height);
            print("Sum² From kernel:\n");
            draw_image_data(
              result_sumsq, mismatch_x, mismatch_y, 16, 16, device_pitch, height);
            print("Mask:\n");
            draw_image_data(reader.get_mask().value().data(),
                            mismatch_x,
                            mismatch_y,
                            16,
                            16,
                            width,
                            height);
        }

        print("\n\n");
    }
}
