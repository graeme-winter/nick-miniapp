#include "hip/hip_runtime.h"
/**
 * Basic Summing with CUDA
 * 
 * Uses h5read to loop over all images, calculates a pixel sum in host
 * and GPU, and compares the results.
 * 
 * Demonstrates using h5read and GPU reduction.
 * 
 */

#include <fmt/core.h>

#include <array>
#include <cassert>
#include <memory>

#include "common.hpp"
#include "h5read.h"
using namespace fmt;

using pixel_t = H5Read::image_type;

template <typename T>
__inline__ __device__ auto warpReduceSum_sync(T val) -> T {
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync((unsigned int)-1, val, offset);
    return val;
}

/// Draw a subset of the pixel values for a 2D image array
/// fast, slow, width, height - describe the bounding box to draw
/// data_width, data_height - describe the full data array size
template <typename T>
void draw_image_data(const T *data,
                     size_t fast,
                     size_t slow,
                     size_t width,
                     size_t height,
                     size_t data_width,
                     size_t data_height) {
    std::string format_type = "";
    if constexpr (std::is_integral<T>::value) {
        format_type = "d";
    } else {
        format_type = ".1f";
    }

    // Walk over the data and get various metadata for generation
    // Maximum value
    T accum = 0;
    // Maximum format width for each column
    std::vector<int> col_widths;
    for (int col = fast; col < fast + width; ++col) {
        size_t maxw = fmt::formatted_size("{}", col);
        for (int row = slow; row < min(slow + height, data_height); ++row) {
            auto val = data[col + data_width * row];
            auto fmt_spec = fmt::format("{{:{}}}", format_type);
            maxw = std::max(maxw, fmt::formatted_size(fmt_spec, val));
            accum = max(accum, val);
        }
        col_widths.push_back(maxw);
    }

    // Draw a row header
    fmt::print("x =       ");
    for (int i = 0; i < width; ++i) {
        auto x = i + fast;
        fmt::print("{:{}} ", x, col_widths[i]);
    }
    fmt::print("\n         ┌");
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < col_widths[i]; ++j) {
            fmt::print("─");
        }
        fmt::print("─");
    }
    fmt::print("┐\n");

    for (int y = slow; y < min(slow + height, data_height); ++y) {
        if (y == slow) {
            fmt::print("y = {:2d} │", y);
        } else {
            fmt::print("    {:2d} │", y);
        }
        for (int i = fast; i < fast + width; ++i) {
            // Calculate color
            // Black, 232->255, White
            // Range of 24 colors, not including white. Split into 25 bins, so
            // that we have a whole black top bin
            // float bin_scale = -25
            auto dat = data[i + data_width * y];
            int color = 255 - ((float)dat / (float)accum) * 24;
            if (color <= 231) color = 0;
            if (dat < 0) {
                color = 9;
            }

            if (dat == accum) {
                fmt::print("\033[0m\033[1m");
            } else {
                fmt::print("\033[38;5;{}m", color);
            }
            auto fmt_spec =
              fmt::format("{{:{}{}}} ", col_widths[i - fast], format_type);
            fmt::print(fmt_spec, dat);
            if (dat == accum) {
                fmt::print("\033[0m");
            }
        }
        fmt::print("\033[0m│\n");
    }
}

__global__ void fill(pixel_t *data, size_t size) {
    for (int i = 0; i < size; ++i) {
        data[i] = 3;
    }
}

/// Print out diagnostics of an image memory area
__global__ void diagnose_memory(pixel_t *data,
                                size_t pitch,
                                size_t width,
                                size_t height) {
    int sum = 0;
    // int calc_width = pitch / sizeof(pixel_t);
    int item_pitch = pitch / sizeof(pixel_t);
    for (int y = 0; y < height; ++y) {
        int last_val = -1;
        int count = 0;
        printf("%4d:  ", y);
        for (int x = 0; x < item_pitch; ++x) {
            if (x == width) {
                if (count > 0) {
                    printf("%d×%-4d ", last_val, count);
                    count = 0;
                }
                printf(" | ");
            }
            size_t index = y * item_pitch + x;
            pixel_t val = data[index];
            sum += val;
            if (val != last_val) {
                if (count > 0) {
                    printf("%d×%-4d ", last_val, count);
                }
                if (val == 3) {
                    printf(">%d< ", x);
                }
                last_val = val;
                count = 0;
            }
            count += 1;
        }
        if (count > 0) {
            printf("%d×%-4d ", last_val, count);
        }
        printf("\n");
    }
    printf("Total: %d\n", sum);
}

__global__ void do_sum_image(size_t *block_store,
                             pixel_t *data,
                             size_t pitch,
                             size_t width,
                             size_t height) {
    // Store an interim block sum int for every warp.
    // In theory this could be less than 32, because we might not have
    // launched the maximum threads. However, then we would need to
    // calculate and pass shared memory requirements on launch. On all
    // cards max_threads <= 1024 (32 warps), so settings to 32 is safe.
    static __shared__ size_t shared[32];

    int warpId = (threadIdx.x + blockDim.x * threadIdx.y) / warpSize;
    int lane = (threadIdx.x + blockDim.x * threadIdx.y) % warpSize;

    // Which position on the image do we need to read
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Always run the warp lane, even if out of image range. This is
    // because the warp shuffle functions will return the input number
    // if the target source is inactive, which gives us phantom numbers
    // for out-of-range pixels.
    size_t pixel = 0;
    if (x < width && y < height) {
        pixel = data[y * (pitch / sizeof(pixel_t)) + x];
    }
    // Sum the current warp
    size_t sum = warpReduceSum_sync(pixel);
    // And save the warp-total to shared memory
    if (lane == 0) {
        shared[warpId] = sum;
    }

    // Wait for all thread warps in the block to write
    __syncthreads();

    if (warpId == 0) {
        // Load the shared memory value for the warp corresponding to this lane.
        // We need to check this, because although we have a maximum number
        // of warps per block (32), we might have had less than that.
        sum = (lane < (blockDim.x * blockDim.y) / warpSize) ? shared[lane] : 0;
        // And sum all of the warps in this block together
        sum = warpReduceSum_sync(sum);
        // Finally, store the block total sum, once.
        if (lane == 0) {
            int blockId = blockIdx.x + gridDim.x * blockIdx.y;
            block_store[blockId] = sum;
        }
    }
}

int main(int argc, char **argv) {
    // Parse arguments and get our H5Reader
    auto parser = CUDAArgumentParser();
    parser.add_h5read_arguments();
    auto args = parser.parse_args(argc, argv);

    auto reader = args.file.empty() ? H5Read() : H5Read(args.file);

    int height = reader.image_shape()[0];
    int width = reader.image_shape()[1];

    // Work out how many blocks this is
    dim3 thread_block_size{32, 16};
    assert(thread_block_size.x == 32);
    dim3 blocks_dims{
      static_cast<unsigned int>(ceilf((float)width / thread_block_size.x)),
      static_cast<unsigned int>(ceilf((float)height / thread_block_size.y))};
    const int num_threads_per_block = thread_block_size.x * thread_block_size.y;
    const int num_blocks = blocks_dims.x * blocks_dims.y;
    print("Image:   {:4d} x {:4d} = {} px\n", width, height, width * height);
    print("Threads: {:4d} x {:<4d} = {}\n",
          thread_block_size.x,
          thread_block_size.y,
          num_threads_per_block);
    print("Blocks:  {:4d} x {:<4d} = {}\n", blocks_dims.x, blocks_dims.y, num_blocks);

    // Create a host memory area to store the current image
    auto host_image = std::make_unique<pixel_t[]>(width * height);

    // Create a device-side pitched area
    pixel_t *dev_image = nullptr;
    size_t device_pitch = 0;
    hipMallocPitch(&dev_image, &device_pitch, width * sizeof(pixel_t), height);
    print("Allocated device memory. Pitch = {} vs naive {}\n",
          device_pitch,
          width * sizeof(pixel_t));
    cuda_throw_error();

    // And a device-side location to store results
    size_t *dev_result = nullptr;
    hipMalloc(&dev_result, sizeof(decltype(*dev_result)) * num_blocks);
    cuda_throw_error();

    for (size_t image_id = 0; image_id < reader.get_number_of_images(); ++image_id) {
        print("Image {}:\n", image_id);
        reader.get_image_into(image_id, host_image.get());

        // Calculate the sum of all pixels host-side
        size_t sum = 0;
        for (int y = 0; y < height; ++y) {
            for (int x = 0; x < width; ++x) {
                sum += host_image[x + y * width];
            }
        }
        print("    Summed pixels: {}\n", bold(sum));

        hipMemcpy2D(dev_image,
                     device_pitch,
                     host_image.get(),
                     width * sizeof(pixel_t),
                     width * sizeof(pixel_t),
                     height,
                     hipMemcpyHostToDevice);
        cuda_throw_error();

        do_sum_image<<<blocks_dims, thread_block_size>>>(
          dev_result, dev_image, device_pitch, width, height);

        hipDeviceSynchronize();
        cuda_throw_error();

        // Copy the per-block sum data back, to sum CPU-side for now
        auto host_result =
          std::make_unique<std::remove_reference<decltype(*dev_result)>::type[]>(
            num_blocks);
        hipMemcpy(host_result.get(),
                   dev_result,
                   sizeof(decltype(*dev_result)) * num_blocks,
                   hipMemcpyDeviceToHost);
        cuda_throw_error();
        hipDeviceSynchronize();

        // Manually sum the response here
        size_t accum = 0;
        for (int i = 0; i < num_blocks; ++i) {
            accum += host_result[i];
        }
        if (accum == sum) {
            print("    Kernel Summed: {}\n", green(bold(accum)));
        } else {
            print("    Kernel Summed: {}\n", red(bold(accum)));
        }

        print("\n");
    }
    hipFree(dev_result);
    hipFree(dev_image);
}
